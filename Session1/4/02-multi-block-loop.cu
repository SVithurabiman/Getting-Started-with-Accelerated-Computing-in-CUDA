
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop(int num_threads)
{
    int i = blockIdx.x* blockDim.x+threadIdx.x;
    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("This is iteration number %d\n", i);

}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, be sure to use more than 1 block in
   * the execution configuration.
   */
  int N=20;
  int num_blocks = 5;
  int num_threads=N/num_blocks;
  loop<<<num_blocks,num_threads>>>(num_threads);
  hipDeviceSynchronize();
}
