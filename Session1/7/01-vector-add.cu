
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initWith(float num, float *a, int N)
{
  int i=  blockIdx.x * blockDim.x + threadIdx.x;
  if(i < N)
  {
    a[i] = num;
  }
}

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
  int i=  blockIdx.x * blockDim.x + threadIdx.x;
  if( i < N)
  {
    result[i] = a[i] + b[i];
  }
}


void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {

    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}


int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  // a = (float *)malloc(size);
  // b = (float *)malloc(size);
  // c = (float *)malloc(size);
  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
  int threads_per_block=1024;
  int number_of_blocks= (N + threads_per_block - 1) / threads_per_block;
  
  initWith<<<number_of_blocks,threads_per_block>>>(3, a, N);
  initWith<<<number_of_blocks,threads_per_block>>>(4, b, N);
  initWith<<<number_of_blocks,threads_per_block>>>(0, c, N);

  addVectorsInto<<<number_of_blocks,threads_per_block>>>(c, a, b, N);

  hipDeviceSynchronize();
  
  checkElementsAre(7, c, N);
  
  hipFree(a);
  hipFree(b);
  hipFree(c);
}
