
#include <hip/hip_runtime.h>
#include <stdio.h>

int main()
{
  /*
   * Assign values to these variables so that the output string below prints the
   * requested properties of the currently active GPU.
   */

  int deviceId;
  int computeCapabilityMajor;
  int computeCapabilityMinor;
  int multiProcessorCount;
  int warpSize;

  /*
   * There should be no need to modify the output string below.
   */
  // int deviceId;
  hipGetDevice(&deviceId);                  // `deviceId` now points to the id of the currently active GPU.

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);
  computeCapabilityMajor=props.major;
  computeCapabilityMinor=props.minor;
  multiProcessorCount=props.multiProcessorCount;
  warpSize = props.warpSize;
  printf("Device ID: %d\nNumber of SMs: %d\nCompute Capability Major: %d\nCompute Capability Minor: %d\nWarp Size: %d\n", deviceId, multiProcessorCount, computeCapabilityMajor, computeCapabilityMinor, warpSize);
}
