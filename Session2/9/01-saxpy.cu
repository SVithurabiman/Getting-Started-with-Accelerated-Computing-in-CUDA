#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Optimize this already-accelerated codebase. Work iteratively,
 * and use nsys to support your work.
 *
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 200,000 ns.
 *
 * Some bugs have been placed in this codebase for your edification.
 */
// __global__ void initWith (int *a,int *b,int *c)
// {

//  int idx = blockIdx.x * blockDim.x * threadIdx.x;
//  int gridStride=gridDim.x*blockDim.x;

//  for (int i=idx; i<N; i+=gridStride){
//   a[i]=2;
//   b[i]=1;
//   c[i]=0;

//  }
// }
__global__ void saxpy(int * a, int * b, int * c)
{
    int tid = blockIdx.x * blockDim.x * threadIdx.x;
    int gridStride=gridDim.x*blockDim.x;
    for (int i=tid; i<N; i+=gridStride){
        c[i] = 2 * a[i] + b[i];
}
}

int main()
{
    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);

    // Initialize memory
    for( int i = 0; i < N; ++i )
    {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }

    int deviceId;
    hipGetDevice(&deviceId);
    int numberOfSMs;
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    hipMemPrefetchAsync(a, size,deviceId);
    hipMemPrefetchAsync(b, size,deviceId);
    hipMemPrefetchAsync(c, size,deviceId);

    int threads_per_block = 1024;
    int number_of_blocks = 32*numberOfSMs;

    // initWith <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    hipDeviceSynchronize();

    // hipMemPrefetchAsync(c, size,hipCpuDeviceId);

    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}
